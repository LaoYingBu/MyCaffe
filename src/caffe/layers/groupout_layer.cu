#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/groupout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void GroupoutForward(const int n, const int group_dim, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index / group_dim] > threshold) * scale;
  }
}

template <typename Dtype>
void GroupoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
	const int group_num = bottom[0]->count(0, 2);
	const int group_dim = bottom[0]->count(2);
	const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
		caffe_gpu_rng_uniform(group_num, mask);
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    GroupoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, group_dim, bottom_data, mask, uint_thres_, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void GroupoutBackward(const int n, const int group_dim, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index / group_dim] > threshold);
  }
}

template <typename Dtype>
void GroupoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
			const int group_num = bottom[0]->count(0, 2);
			const int group_dim = bottom[0]->count(2);
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      GroupoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, group_dim, top_diff, mask, uint_thres_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GroupoutLayer);

}  // namespace caffe
