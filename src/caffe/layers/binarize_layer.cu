#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/binarize_layer.hpp"

namespace caffe {

template <typename Dtype>
void BinarizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
	caffe_gpu_sign(count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void BinarizeBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
		//Dtype tanhx = tanh(in_data[index]);
		out_diff[index] = in_diff[index] * (1 - abs(in_data[index]));
  }
}

template <typename Dtype>
void BinarizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    BinarizeBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BinarizeLayer);


}  // namespace caffe
